#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sum_slice/cuda.cu
 * Sums over fibers into a slice of a buffer on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/sum_slice/cuda.hh"
#include <algorithm>

namespace nntile::kernel::sum_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, T alpha, const T *src,
        T beta, T *dst)
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y;
    Index i2_start = threadIdx.z, i2_step = blockDim.z;
    constexpr T zero = 0;
    if(i0 < m and i1 < n)
    {
        // Pointer to a corresponding fiber of the source array src
        const T *src_fiber = src + i1*mk + i0;
        // Init sum over the fiber
        T sum = zero;
        // Cycle over fiber elements and accumulate the sum
        for(Index i2 = i2_start; i2 < k; i2 += i2_step)
        {
            sum += src_fiber[i2*m];
        }
        __shared__ T block_sum[64];
        if(i2_start == 0)
        {
            block_sum[threadIdx.x+blockDim.x*threadIdx.y] = zero;
        }
        __syncthreads();
        atomicAdd(&block_sum[threadIdx.x+blockDim.x*threadIdx.y], sum);
        __syncthreads();
        // Update output value
        if(i2_start == 0)
        {
            // Output value
            T &result = dst[i1*m+i0];
            sum = block_sum[threadIdx.x+blockDim.x*threadIdx.y];
            if(beta == zero)
            {
                result = alpha * sum;
            }
            else
            {
                result = beta*result + alpha*sum;
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, T alpha,
        const T *src, T beta, T *dst)
    noexcept
//! Sums over fibers along middle axis into a slice of a tensor
/*! For a provided m-by-k-by-n input array computes sums over fibers
 * along second axis with k elements, resulting in m-by-n output slice.
 * Mnemonically, the following operations are performed:
 *      dst[i,j] = beta*dst[i,j] + alpha*sum(src[i,:,j])
 *
 * @param[in] m: Size of the first mode of src and dst arrays
 * @param[in] n: Size of the last mode of src and dst arrays
 * @param[in] k: Size of the middle mode of src array
 * @param[in] alpha: Scaling factor for src
 * @param[in] src: Input contiguous m-by-k-by-n array
 * @param[in] beta: Scaling factor for dst
 * @param[inout] sum: Output contiguous m-by-n array, that accumulates
 *      sums over fibers along middle axis.
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y, 1);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha, src,
            beta, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, fp32_t alpha,
        const fp32_t *src, fp32_t beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, fp64_t alpha,
        const fp64_t *src, fp64_t beta, fp64_t *dst)
    noexcept;

} // namespace nntile::kernel::sum_slice

